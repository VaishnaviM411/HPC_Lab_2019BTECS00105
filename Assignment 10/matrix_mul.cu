#include "hip/hip_runtime.h"
#include <stdio.h>

void initWith(float num, float *a, int SIZE)
{
  for(int i = 0; i < SIZE; ++i)
  {
    a[i] = num;
  }
}

void matrixMultiply(float *result, float *a, float *b, int N, int SIZE)
{

  for(int i = 0; i < SIZE; i ++)
  {
    int row = i / N;

    float sum = 0;

    for (int j = 0; j < N; j++)
    {
      sum += a[row * N + j] * b[N * j + row];
    }

    result[i] = sum;
  }
}

void checkElementsAre(float target, float *array, int SIZE)
{
  for(int i = 0; i < SIZE; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values multiplied correctly.\n");
}

int main()
{
  const int N = 1024;
  const int SIZE = N * N; // sqaure matrix
  //size_t size = SIZE * sizeof(float);

  float *a=(float*)malloc(N*N * sizeof(float));
  float *b=(float*)malloc(N*N * sizeof(float));
  float *c=(float*)malloc(N*N * sizeof(float));

  initWith(3, a, SIZE);
  initWith(4, b, SIZE);
  initWith(0, c, SIZE);

  double time_spent = 0.0;
  clock_t begin = clock();
  matrixMultiply(c, a, b, N, SIZE);

  clock_t end = clock();
 
  time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
  printf("The elapsed time is %f seconds", time_spent);
 

  checkElementsAre(12288, c, SIZE);

}
