#include "hip/hip_runtime.h"
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

void addVectorsInto(float *result, float *a, float *b, int N)
{
  for(int i = 0; i < N; i ++)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;

  float *a=(float*)malloc(N * sizeof(float));
  float *b=(float*)malloc(N * sizeof(float));
  float *c=(float*)malloc(N * sizeof(float));

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  double time_spent = 0.0;
  clock_t begin = clock();

  addVectorsInto(c, a, b, N);

  clock_t end = clock();
 
  time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
  printf("The elapsed time is %f seconds", time_spent);
 
  checkElementsAre(7, c, N);
}
