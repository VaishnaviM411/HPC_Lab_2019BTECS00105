#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  checkCuda( hipMallocManaged(&a, size) );
  checkCuda( hipMallocManaged(&b, size) );
  checkCuda( hipMallocManaged(&c, size) );

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 1024;
  numberOfBlocks = 25;

  double time_spent = 0.0;
  clock_t begin = clock();

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  clock_t end = clock();
 
  time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
  printf("The elapsed time is %f seconds", time_spent);
 
  checkCuda( hipGetLastError() );
  checkCuda( hipDeviceSynchronize() );

  checkElementsAre(7, c, N);

  checkCuda( hipFree(a) );
  checkCuda( hipFree(b) );
  checkCuda( hipFree(c) );
}
